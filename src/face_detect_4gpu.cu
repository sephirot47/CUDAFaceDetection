#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../include/stbi.h"
#include "../include/stbi_write.h"

#define IMG_WIDTH 1024
#define IMG_HEIGHT 1024
#define NUM_THREADS 1024
#define NUM_BLOCKS 255
#define IMG_CHANNELS 3
#ifndef NUM_DEVICES
#define NUM_DEVICES 4
#endif

// Optimal values 40, 550
#define THRESH_9x9 20     // Greater = more restrictive
#define THRESH_30x30 475  // Greater = less restrictive

typedef unsigned char uc;
using namespace std;

struct Pixel { int x,y; Pixel(int _x, int _y) {x=_x; y=_y;} };

struct Color { int r,g,b,a; Color(int _r, int _g, int _b, int _a) { r=_r; g=_g; b=_b; a=_a; } };

class Image
{
public:
    uc *data;
    char* filename;

    Image(char* _filename) {
	filename = _filename;
	FILE *file = fopen(_filename, "r");
	if (file != NULL) {
	    data = stbi_load_from_file(file, &_width, &_height, &comp, IMG_CHANNELS); // rgb
	    fclose (file);
	    printf("%s read successfully\n", _filename);
	}
	else printf("Image '%s' not found\n", _filename);
    }

    int width() { return _width;  }
    int height() { return _height; }

    Color getColor(Pixel p) {
	int offset = (p.y * _width + p.x) * comp;
	return Color((data[offset + 0]), (data[offset + 1]),
		     (data[offset + 2]), (data[offset + 3]));
    }

    void setColor(Pixel p, Color c) {
	int offset = (p.y * _width + p.x) * 3;
	data[offset + 0] = c.r;
	data[offset + 1] = c.g;
	data[offset + 2] = c.b;
    }

    uc getGrayScale(Pixel p) { return grayscale(getColor(p)); }

private:
    int _width, _height, comp;
    uc grayscale(const Color &c) { return 0.299*c.r + 0.587*c.g + 0.114*c.b; }
};

struct Box { int x,y,w,h; Box(int _x, int _y, int _w, int _h) { x=_x; y=_y; w=_w; h=_h; } };

class FaceDetection
{
public:
    vector<Box> resultWindows;
    Image *image;

    FaceDetection(char* imageFile)
    {
	image = new Image(imageFile);
    }

    void saveResult() {
	printf("Saving result...\n");

	Color c = Color(0,255,0,255);
	for(Box b : resultWindows) {
	    for (int i = b.x; i < b.x+b.w; ++i) {
		image->setColor(Pixel(i,b.y),c);
		image->setColor(Pixel(i,b.y+1),c);
	    }
	    for (int i = b.x; i < b.x+b.w; ++i) {
		image->setColor(Pixel(i,b.y+b.h-1),c);
		image->setColor(Pixel(i,b.y+b.h-2),c);
	    }
	    for (int i = b.y; i < b.y+b.h; ++i) {
		image->setColor(Pixel(b.x,i),c);
		image->setColor(Pixel(b.x+1,i),c);
	    }
	    for (int i = b.y; i < b.y+b.h; ++i) {
		image->setColor(Pixel(b.x+b.w-1,i),c);
		image->setColor(Pixel(b.x+b.w-2,i),c);
	    }
	}

	stbi_write_bmp("output/result.bmp", image->width(), image->height(), 3, image->data);
    }

};

void saveImage(uc *img, int x, int y, int width, int height, int imgWidth, const char *filename)
{
    uc *aux = (uc*) malloc(width * height * 3 * sizeof(uc));
    for(int iy = 0; iy < height; ++iy)
    {
        for(int ix = 0; ix < width; ++ix)
        {
            int offset = (iy * width + ix) * 3;
            aux[offset + 0] = img[(iy+y) * imgWidth + ix + x];
            aux[offset + 1] = img[(iy+y) * imgWidth + ix + x];
            aux[offset + 2] = img[(iy+y) * imgWidth + ix + x];
        }
    }
    stbi_write_bmp(filename, width, height, 3, aux);
    free(aux);
}

__device__ __host__ uc getWindowMeanGS(uc *img, int ox, int oy, int winWidth, int winHeight, int imgWidth) {
    int sum = 0;
    for(int y = oy; y < oy + winHeight; ++y)
    {
        for(int x = ox; x < ox + winWidth; ++x)
        {
            int offset = y * imgWidth + x;
            sum += img[offset];
        }
    }

    return uc(sum / (winWidth * winHeight));
}

void resize_seq(uc *src, int srcx, int srcy, int srcw, int srch, int srcTotalWidth, //x,y,width,height
                uc *dst, int dstx, int dsty, int dstw, int dsth, int dstTotalWidth) //x,y,width,height
{
    //Every square of size (bw,bh), will be substituted
    //by one pixel in the dst image
    float bw = float(srcw) / dstw;
    float bh = float(srch) / dsth;

    //For each pixel in the dst
    for(int dy = dsty; dy < dsty + dsth; ++dy)
    {
        for(int dx = dstx; dx < dstx + dstw; ++dx)
        {
            //Save in its position the mean of the corresponding window pixels
            uc mean = getWindowMeanGS(src,
                                      srcx + ceil(dx*bw), srcy + ceil(dy*bh), //x, y
                                      floor(bw), floor(bh),                   //width height
                                      srcTotalWidth                           //totalWidth
                                      );

            dst[dy * dstTotalWidth + dx] = mean;
        }
    }
}

// Resize always to a smaller size -> downsample
__device__ void resize(uc *src, int srcx, int srcy, int srcw, int srch, int srcTotalWidth, //x,y,width,height
                       uc *dst, int dstx, int dsty, int dstw, int dsth, int dstTotalWidth) //x,y,width,height
{
    float bw = float(srcw) / dstw;
    float bh = float(srch) / dsth;

    int size = dsth * dstw;
    for(int i = threadIdx.x; i < size; i += NUM_THREADS)
    {
        int dx = dstx + (i % dstw);
        int dy = dsty + (i / dstw);

        uc mean = getWindowMeanGS(src,
                                  srcx + ceil(dx*bw), srcy + ceil(dy*bh), //x, y
                                  floor(bw), floor(bh),                   //width height
                                  srcTotalWidth                           //totalWidth
                                  );

        dst[dy * dstTotalWidth + dx] = mean;
    }
}


__device__ void getHistogram(uc *img, int ox, int oy, int width, int height, int imgWidth, float histogram[256]) {

    float npixels = width * height;
    float unitProb = 1.0f/npixels;

    if(threadIdx.x < 256) histogram[threadIdx.x] = 0;

    for(int i = threadIdx.x; i < npixels; i += NUM_THREADS)
    {
        int wx = i % width;
        int wy = i / width;
        int offset = (oy + wy) * imgWidth + (ox + wx);
        uc v = img[offset];
        atomicAdd(&histogram[v], unitProb);
    }
}

// Increase contrast
__device__ void histogramEqualization(uc *img, int ox, int oy, int width, int height, int imgWidth)
{
    __shared__ float histogram[256];
    __shared__ float accumulatedProbs[256];
    
    getHistogram(img, ox, oy, width, height, imgWidth, histogram);
    
    if(threadIdx.x == 0)
    { 
        accumulatedProbs[0] = histogram[0];
    	for(int i = 1; i < 256; ++i)
            accumulatedProbs[i] = accumulatedProbs[i-1] + histogram[i];
    }
    __syncthreads();
    
    
    int size = width * height;
    for(int i = threadIdx.x; i < size; i += NUM_THREADS)
    {
	 int wx = i % width;
	 int wy = i / width;
	 int offset = (oy + wy) * imgWidth + (ox + wx);
    	 uc v = img[offset];
         img[offset] = floor(255 * accumulatedProbs[v]);
    }
}

__device__ void toBlackAndWhite(uc *img, int ox, int oy, int width, int height, int imgWidth)
{
    int size = width * height;
    for(int i = threadIdx.x; i < size; i += NUM_THREADS)
    {
        int wx = i % width;
        int wy = i / width;
        int offset = (oy + wy) * width + (ox + wx);
        uc v = img[offset];
        img[offset] = v > 200 ? 255 : 0;
    }
}

__device__ uc getFirstStageHeuristic(uc *img) {
    int v = img[22] - (img[19]+img[20]+img[24]+img[25]+img[58])/5;
    return v < 0 ? 0 : v;
}

// Find edges in horizontal direction
__device__ void sobelEdgeDetection(uc *img, int ox, int oy,  int winWidth, int winHeight, int imgWidth, uc *sobelImg)
{
    uc threshold = 24;

    int size = winWidth * winHeight;
    for (int i = threadIdx.x; i < size; i+=NUM_THREADS)
    {
        int wx = i % winWidth;
        int wy = i / winWidth;
        int winOffset = wy * winWidth + wx;

        int x = ox + wx;
        int y = oy + wy;
        int imgOffset = y * imgWidth + x;

        if (y == oy or y == oy+winHeight-1 or x == ox or x == ox+winWidth-1)
            sobelImg[winOffset] = 255;
        else {
            uc upperLeft  = img[imgOffset - imgWidth - 1];
            uc upperRight = img[imgOffset - imgWidth + 1];
            uc up         = img[imgOffset - imgWidth];
            uc down       = img[imgOffset + imgWidth];
            uc lowerLeft  = img[imgOffset + imgWidth - 1];
            uc lowerRight = img[imgOffset + imgWidth + 1];

            int sum = -upperLeft - upperRight - 2*up + 2*down + lowerLeft + lowerRight;
            if(sum >= threshold)
                sobelImg[winOffset] = 0;
            else
                sobelImg[winOffset] = 255;
        }
    }
}

__device__ int getSecondStageHeuristic(uc *img) {
    int sumDiff    = 0;
    int leftEye    = getWindowMeanGS(img, 2, 4, 9, 5,30);
    int rightEye   = getWindowMeanGS(img,18, 4, 9, 5,30);
    int upperNose  = getWindowMeanGS(img,11, 1, 6,13,30);
    int lowerNose  = getWindowMeanGS(img,10,15, 9, 5,30);
    int leftCheek  = getWindowMeanGS(img, 1,10, 8,10,30);
    int rightCheek = getWindowMeanGS(img,19,10, 8,10,30);
    int mouth      = getWindowMeanGS(img, 8,21,13, 5,30);

    sumDiff += leftEye;
    sumDiff += rightEye;
    sumDiff += abs(leftEye - rightEye); // simmetry

    sumDiff += 255-upperNose;
    sumDiff += abs(125-lowerNose);

    sumDiff += 255-leftCheek;
    sumDiff += 255-rightCheek;
    sumDiff += abs(leftCheek - rightCheek); // simmetry

    sumDiff += mouth; // mouth

    return sumDiff;
}

__global__ void detectFaces(uc *img, int winWidth, int winHeight, uc  *resultMatrix)
{
    int xstep = (IMG_WIDTH - winWidth) / NUM_BLOCKS + 1;
    int ystep = (IMG_HEIGHT - winHeight) / NUM_BLOCKS + 1;
    if(threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    {
    	printf("Kernel width:%i, height:%i\n", winWidth, winHeight);
        printf("steps: %d, %d\n", xstep, ystep);
    }

    // Window origin
    int x = blockIdx.x * xstep;
    int y = blockIdx.y * ystep;
    int blockId = blockIdx.y * NUM_BLOCKS + blockIdx.x;

    if(x + winWidth > IMG_WIDTH || y + winHeight > IMG_HEIGHT)
    {
        resultMatrix[blockId] = 0;
        return;
    }

    // FIRST HEURISTIC
    __shared__ uc window30x30[30*30];
    resize(img,
           x, y, winWidth, winHeight, IMG_WIDTH,
           window30x30,
           0, 0, 9, 9, 9);
    __syncthreads();

    histogramEqualization(window30x30, 0, 0, 9, 9, 9);
    __syncthreads();
    
    __shared__ uc hv1;
    if(threadIdx.x == 0) {
        hv1 = getFirstStageHeuristic(window30x30);
    }
    __syncthreads();

    if (hv1 >= THRESH_9x9)
    {
        // SECOND HEURISTIC
        __shared__ uc sobelImg[200*200];
        sobelEdgeDetection(img, x, y, winWidth, winHeight, IMG_WIDTH, sobelImg);
        __syncthreads();

	resize(sobelImg,
               0, 0, winWidth, winHeight, winWidth,
               window30x30,
               0, 0, 30, 30, 30);
        __syncthreads();

        toBlackAndWhite(window30x30, 0, 0, 30, 30, 30);
        __syncthreads();

        if(threadIdx.x == 0) {
            int hv2 = getSecondStageHeuristic(window30x30);

            if (hv2 <= THRESH_30x30)
            {
		printf("Face detected with heuristic: %i\n", hv2);
                // Save result! We detected a face yayy
                resultMatrix[blockId] = 1;
            }
            else resultMatrix[blockId] = 0;
        }
    }
    else resultMatrix[blockId] = 0;
}

void CheckCudaError(int line) {
    hipError_t error;
    error = hipGetLastError();
    if (error) {
	printf("(ERROR) - %s in %s at line %d\n", hipGetErrorString(error), __FILE__, line);
	exit(EXIT_FAILURE);
    }
}

#define CE() { CheckCudaError(__LINE__); }

int main(int argc, char** argv)
{
    cout << "Usage: " << argv[0] << " <image file name>" << endl;
    for (int i = 0; i < argc; ++i) { cout << argv[i] << endl; }

    //Read input
    FaceDetection fc(argv[1]);
    printf("image File: %s, size(%d px, %d px)\n",
	    fc.image->filename, fc.image->width(), fc.image->height());

    //Adapt input
    int numBytesImageOriginal = fc.image->width() * fc.image->height() * sizeof(uc);
    uc *h_imageGSOriginal = (uc*) malloc(numBytesImageOriginal);
    printf("Adapting input. Creating grayscale image....\n");
    for(int y = 0; y < fc.image->height(); ++y) {
	for(int x = 0; x < fc.image->width(); ++x) {
	    h_imageGSOriginal[y * fc.image->width() + x] = fc.image->getGrayScale(Pixel(x,y));
	}
    }

    printf("Resizing original image....\n");
    int numBytesImage = IMG_WIDTH * IMG_HEIGHT * sizeof(uc);
    uc *h_imageGS = (uc*) malloc(numBytesImage);
    resize_seq(h_imageGSOriginal,
	       0, 0, fc.image->width(), fc.image->height(), fc.image->width(),
	       h_imageGS,
	       0, 0, IMG_WIDTH, IMG_HEIGHT, IMG_WIDTH);

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < NUM_DEVICES) { printf("Not enough GPUs\n"); exit(-1); }

    // Get device memory
    dim3 dimGrid(NUM_BLOCKS, NUM_BLOCKS, 1);
    dim3 dimBlock(NUM_THREADS, 1, 1);
    int winWidths[] = {35, 40, 45, 50, 55, 60, 65}; //, 75, 85, 95, 105, 115, 125, 140, 150, 160, 170, 180, 190};
    int winHeights[] = {35, 40, 45, 50, 55, 60, 65}; //, 75, 85, 95, 105, 115, 125, 140, 150, 160, 170, 180, 190};

    printf("Getting memory in the host to allocate resultMatrix...\n");
    int numBytesResultMatrix = NUM_BLOCKS * NUM_BLOCKS * sizeof(uc);

    const int numWindowsWidth  = (sizeof(winWidths) / sizeof(int));
    const int numWindowsHeight = (sizeof(winHeights) / sizeof(int));
    const int numWindows = numWindowsWidth * numWindowsHeight;
    uc *h_resultMatrix[numWindows];
    for(int i = 0; i < numWindows; ++i)
	h_resultMatrix[i]= (uc*) malloc(numBytesResultMatrix);

    uc *d_imageGS[NUM_DEVICES], *d_resultMatrix[NUM_DEVICES];
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
	printf("Getting memory in device %d...\n", i);
	hipSetDevice(i);
	#ifndef USE_PINNED
	hipMalloc((uc**)&d_imageGS[i], numBytesImage); CE();
	hipMalloc((uc**)&d_resultMatrix[i], numBytesResultMatrix); CE();
	#else
	hipHostMalloc((uc**)&d_imageGS[i], numBytesImage); CE();
	hipHostMalloc((uc**)&d_resultMatrix[i], numBytesResultMatrix); CE();
	#endif
    }

    // Copy data from host to device, execute kernel, copy data from device to host
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
	printf("Getting memory in device %d...\n", i);
	hipSetDevice(i);
	#ifndef USE_PINNED
	hipMalloc((uc**)&d_imageGS[i], numBytesImage); CE();
	hipMalloc((uc**)&d_resultMatrix[i], numBytesResultMatrix); CE();
	#else
	hipHostMalloc((uc**)&d_imageGS[i], numBytesImage); CE();
	hipHostMalloc((uc**)&d_resultMatrix[i], numBytesResultMatrix); CE();
	#endif
    }

    hipEvent_t E0, E1;
    hipEventCreate(&E0);
    hipEventCreate(&E1);

    float widthRatio =  float(fc.image->width())/IMG_WIDTH;
    float heightRatio =  float(fc.image->height())/IMG_HEIGHT;
    const int windowsPerDevice = numWindows / NUM_DEVICES;

    hipEventRecord(E0, 0);
    hipEventSynchronize(E0);

    printf("wr: %f\n", widthRatio);
    printf("hr: %f\n", heightRatio);
    printf("Num windows: %i\n", numWindows);
    printf("Windows per device: %i\n", windowsPerDevice);
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
	hipSetDevice(i);
	for(int j = 0; j < windowsPerDevice; ++j)
	{
	    int index = (i*windowsPerDevice + j);
	    int wi = index / numWindowsWidth;
	    int hi = index % numWindowsHeight;
	    printf("\n");
	    printf("index: %i\n", index);
	    printf("wi: %i, hi: %i\n", wi, hi);
	    printf("width: %i, height:%i\n", winWidths[wi], winHeights[hi]);
	    printf("Copying matrices from host to device %d...\n", i);
	    hipMemcpyAsync(d_imageGS[i], h_imageGS, numBytesImage, hipMemcpyHostToDevice); //CE();
	    printf("Executing kernel detectFaces on device %d...\n", i);
	    detectFaces<<<dimGrid, dimBlock>>>(d_imageGS[i], winWidths[wi] / widthRatio, winHeights[hi] / heightRatio, d_resultMatrix[i]); //CE();
	    printf("Retrieving resultMatrix from device %d to host...\n", i);
	    hipMemcpyAsync(h_resultMatrix[index], d_resultMatrix[i], numBytesResultMatrix, hipMemcpyDeviceToHost); //CE();
	}
    }

    for(int i = 0; i < NUM_DEVICES; ++i) { hipSetDevice(i); hipDeviceSynchronize(); }

    hipEventRecord(E1, 0);
    hipEventSynchronize(E1);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,  E0, E1);
    printf("Kernel elapsed time: %4.6f\n", elapsedTime);

    // Process results
    for(int k = 0; k < numWindows; ++k)
    {
    for(int i = 0; i < NUM_BLOCKS; ++i)
    {
	for(int j = 0; j < NUM_BLOCKS; ++j)
	{
	    if (h_resultMatrix[k][i * NUM_BLOCKS + j] == 1) 
	    {
		int wi = k / numWindowsWidth;
		int hi = k % numWindowsHeight;
		int kernelStepWidth = (IMG_WIDTH - winWidths[wi]/widthRatio) / NUM_BLOCKS + 1;
		int kernelStepHeight = (IMG_HEIGHT - winHeights[hi]/heightRatio) / NUM_BLOCKS + 1;
		printf("Result found for size(%d,%d) in x,y: (%d,%d)\n", winWidths[wi], winHeights[hi], j, i);
		fc.resultWindows.push_back(Box(int(j * kernelStepWidth * widthRatio),
					       int(i * kernelStepHeight * heightRatio),
					       int(winWidths[wi]),
					       int(winHeights[hi])));
	    }
	}
    }
    }
    fc.saveResult();

    // Free device memory
    printf("Freeing device memory...\n");
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
	hipSetDevice(i); 
	hipFree(d_imageGS[i]); 
	hipFree(d_resultMatrix[i]);
    }

    printf("Done.");
}


