#include "hip/hip_runtime.h"
#include "common.h"
#include "kernels.cu"

void resize_seq(uc *src, int srcx, int srcy, int srcw, int srch, int srcTotalWidth, //x,y,width,height
                uc *dst, int dstx, int dsty, int dstw, int dsth, int dstTotalWidth) //x,y,width,height
{
    //Every square of size (bw,bh), will be substituted
    //by one pixel in the dst image
    float bw = float(srcw) / dstw;
    float bh = float(srch) / dsth;

    //For each pixel in the dst
    for(int dy = dsty; dy < dsty + dsth; ++dy)
    {
        for(int dx = dstx; dx < dstx + dstw; ++dx)
        {
            //Save in its position the mean of the corresponding window pixels
            uc mean = getWindowMeanGS(src,
                                      srcx + ceil(dx*bw), srcy + ceil(dy*bh), //x, y
                                      floor(bw), floor(bh),                   //width height
                                      srcTotalWidth                           //totalWidth
                                      );

            dst[dy * dstTotalWidth + dx] = mean;
        }
    }
}

__global__ void detectFaces(uc *img, int winWidth, int winHeight, uc  *resultMatrix, int resultIndex);

void CheckCudaError(int line) {
    hipError_t error;
    error = hipGetLastError();
    if (error) {
        printf("(ERROR) - %s in %s at line %d\n", hipGetErrorString(error), __FILE__, line);
        exit(EXIT_FAILURE);
    }
}

#define CE() { CheckCudaError(__LINE__); }

int main(int argc, char** argv)
{
    cout << "Usage: " << argv[0] << " <image file name>" << endl;
    for (int i = 0; i < argc; ++i) { cout << argv[i] << endl; }

    // Read input image
    FaceDetection fc(argv[1]);
    printf("image File: %s, size(%d px, %d px)\n",
           fc.image->filename, fc.image->width(), fc.image->height());

    // Convert input image to grayscale
    int numBytesImageOriginal = fc.image->width() * fc.image->height() * sizeof(uc);
    uc *h_imageGSOriginal = (uc*) malloc(numBytesImageOriginal);
    printf("Adapting input. Creating grayscale image....\n");
    for(int y = 0; y < fc.image->height(); ++y) {
        for(int x = 0; x < fc.image->width(); ++x) {
            h_imageGSOriginal[y * fc.image->width() + x] = fc.image->getGrayScale(Pixel(x,y));
        }
    }

    // Resize input image
    printf("Resizing original image....\n");
    int numBytesImage = IMG_WIDTH * IMG_HEIGHT * sizeof(uc);
    uc *h_imageGS = (uc*) malloc(numBytesImage);
    resize_seq(h_imageGSOriginal,
               0, 0, fc.image->width(), fc.image->height(), fc.image->width(),
               h_imageGS,
               0, 0, IMG_WIDTH, IMG_HEIGHT, IMG_WIDTH);


    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < NUM_DEVICES) { printf("Not enough GPUs\n"); exit(-1); }

    dim3 dimGrid(NUM_BLOCKS, NUM_BLOCKS, 1);
    dim3 dimBlock(NUM_THREADS, 1, 1);
    int winWidths[] = {35, 40, 45, 50, 55, 60, 65, 70, 75, 85, 95, 105, 115, 125, 140, 150, 160, 170, 180, 190};
    int winHeights[] = {35, 40, 45, 50, 55, 60, 65, 70, 75, 85, 95, 105, 115, 125, 140, 150, 160, 170, 180, 190};

    const int numWindowWidths  = (sizeof(winWidths) / sizeof(int));
    const int numWindowHeights = (sizeof(winHeights) / sizeof(int));
    const int numWindows = numWindowWidths * numWindowHeights;
    const int windowsPerDevice = numWindows / NUM_DEVICES;
    const float widthRatio =  float(fc.image->width())/IMG_WIDTH;
    const float heightRatio =  float(fc.image->height())/IMG_HEIGHT;

    printf("Num windows: %i\n", numWindows);
    printf("Windows per device: %i\n", windowsPerDevice);


    // Get host memory
    printf("Getting memory in the host to allocate resultMatrix...\n");
    int numBytesResultMatrix = NUM_BLOCKS * NUM_BLOCKS * windowsPerDevice * sizeof(uc);
    uc *h_resultMatrix[NUM_DEVICES];
    for(int i = 0; i < NUM_DEVICES; ++i)
        h_resultMatrix[i] = (uc*) malloc(numBytesResultMatrix);

    // Get memory in every device
    uc *d_imageGS[NUM_DEVICES], *d_resultMatrix[NUM_DEVICES];
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
        printf("Getting memory in device %d...\n", i);
        hipSetDevice(i);
        #ifndef USE_PINNED
        hipMalloc((uc**)&d_imageGS[i], numBytesImage); CE();
        hipMalloc((uc**)&d_resultMatrix[i], numBytesResultMatrix); CE();
        #else
        hipHostMalloc((uc**)&d_imageGS[i], numBytesImage); CE();
        hipHostMalloc((uc**)&d_resultMatrix[i], numBytesResultMatrix); CE();
        #endif
    }


    // Copy data from host to device, execute kernel
    hipSetDevice(0);
    hipEvent_t E0, E1;
    hipEventCreate(&E0);
    hipEventCreate(&E1);

    hipEventRecord(E0, 0);

    for(int i = 0; i < NUM_DEVICES; ++i)
    {
        hipSetDevice(i);
        hipMemcpyAsync(d_imageGS[i], h_imageGS, numBytesImage, hipMemcpyHostToDevice); CE();
        for(int j = 0; j < windowsPerDevice; ++j)
        {
            int index = (i*windowsPerDevice + j);
            int wi = index / numWindowWidths;
            int hi = index % numWindowHeights;
            detectFaces<<<dimGrid, dimBlock>>>(d_imageGS[i],
                                               winWidths[wi] / widthRatio,
                                               winHeights[hi] / heightRatio,
                                               d_resultMatrix[i],
                                               j); CE();
        }
    }

    // Retrieve results from device to host
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
        hipSetDevice(i);
        hipMemcpyAsync(h_resultMatrix[i], d_resultMatrix[i], numBytesResultMatrix, hipMemcpyDeviceToHost);
    }

    for(int i = 0; i < NUM_DEVICES; ++i) {
        hipSetDevice(i);
        hipDeviceSynchronize();
    }

    hipSetDevice(0);
    hipEventRecord(E1, 0);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,  E0, E1);
    printf("Kernel elapsed time: %4.6fs\n", elapsedTime / 1000.0);


    // Process results
    for(int k = 0; k < numWindows; ++k)
    {
    for(int i = 0; i < NUM_BLOCKS; ++i)
    {
    for(int j = 0; j < NUM_BLOCKS; ++j)
    {
        int device = k / windowsPerDevice;
        int resultIndex = k % windowsPerDevice;
        int resultOffset = resultIndex * NUM_BLOCKS * NUM_BLOCKS;
        if (h_resultMatrix[device][i * NUM_BLOCKS + j + resultOffset] == 1)
        {
            int wi = k / numWindowWidths;
            int hi = k % numWindowHeights;
            int kernelStepWidth = (IMG_WIDTH - winWidths[wi]/widthRatio) / NUM_BLOCKS + 1;
            int kernelStepHeight = (IMG_HEIGHT - winHeights[hi]/heightRatio) / NUM_BLOCKS + 1;
            printf("Result found for size(%d,%d) in x,y: (%d,%d)\n", winWidths[wi], winHeights[hi], j, i);
            fc.resultWindows.push_back(Box(int(j * kernelStepWidth * widthRatio),
                                           int(i * kernelStepHeight * heightRatio),
                                           int(winWidths[wi]),
                                           int(winHeights[hi])));
        }
    }
    }
    }
    fc.saveResult();


    // Free memory in every device
    printf("Freeing device memory...\n");
    for(int i = 0; i < NUM_DEVICES; ++i)
    {
    hipSetDevice(i);
    hipFree(d_imageGS[i]);
    hipFree(d_resultMatrix[i]);
    }

    printf("Done.\n");
}
