#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stbi.h"
#include "stbi_write.h"

typedef unsigned char uc;
using namespace std;

class Pixel
{
public:
  int x,y;

  Pixel() { x = y = 0; }
  Pixel(int _x, int _y) {x=_x; y=_y;}

  string toString() {
    ostringstream ss;
    ss << "(" << x << ",\t" << y << ")";
    return ss.str();

  }
};

class Color
{
public:
  uc r,g,b,a;

  Color(uc _r, uc _g, uc _b, uc _a) { r=_r; g=_g; b=_b; a=_a; }

  string toString() {
    ostringstream ss;
    ss << "(" << r << "," << g << "," << b << "," << a << ")";
    return ss.str();
  }
};

class Image
{
public:
  Image(char* filename) {
    FILE *file = fopen(filename, "r");
    if (file != NULL)
    {
      data = stbi_load_from_file(file, &_width, &_height, &comp, 4); // rgba
      fclose (file);
    }
    else
    {
        printf("Image '%s' not found\n", filename);
    }
  }

  int width() {
      return _width;
  }

  int height() {
      return _height;
  }

  Color getColor(Pixel p) {
      int offset = (p.y * _width + p.x) * comp;
      Color c((data[offset + 0]),
              (data[offset + 1]),
              (data[offset + 2]),
              (data[offset + 3]));
      return c;
  }

  uc getGrayScale(Pixel p) {
    return grayscale(getColor(p));
  }

private:
  uc *data;
  int _width, _height, comp;

  uc grayscale(const Color &c) {
      return 0.299*c.r + 0.587*c.g + 0.114*c.b;
  }

};


class FaceDetection
{
public:
  Image *container, *object;

  FaceDetection(char* containerFile, char* objectFile)
  {
    container = new Image(containerFile);
    object = new Image(objectFile);
  }

  // 0:   exact same image
  // 255: very different
  uc getWindowDiff(Pixel origin, int windowWidth, int windowHeight)
  {
    int totalDiff = 0;
    for(int y = 0; y < windowHeight; ++y)
    {
      for(int x = 0; x < windowWidth; ++x)
      {
        Pixel cPixel(origin.x + x, origin.y + y); // container pixel
        Pixel oPixel(x,y);                        // object pixel

        uc cAlpha = container->getColor(cPixel).a;
        uc oAlpha = container->getColor(oPixel).a;
        uc cGray = container->getGrayScale(cPixel);
        uc oGray = object->getGrayScale(oPixel);

        uc diffAlpha = getDiff(cAlpha, oAlpha);
        if(diffAlpha < 200) // if alpha channels differ, ignore the diff (consider pixels are equal)
        {
          uc diff = getDiff(cGray, oGray);
          totalDiff += diff;
        }
      }
    }

    return uc(totalDiff / (windowWidth * windowHeight));
  }


  // 0:   same grayscale
  // 255: very different
  uc getDiff(uc gray1, uc gray2)
  {
    return abs(gray1-gray2);
  }

  void saveResult(Pixel origin) {
      uc *result = new uc[container->width() * container->height() * 3 * sizeof(uc)];
      for(int y = 0; y < container->height(); ++y) {
          for(int x = 0; x < container->width();  ++x) {
              Pixel p(x,y);
              int offset = (y * container->width() + x) * 3;
              if (belongsTo(p,origin,object->width(),object->height())) {
                  result[offset + 0] = container->getColor(p).r;
                  result[offset + 1] = 255;
                  result[offset + 2] = container->getColor(p).b;
              }
              else {
                  result[offset + 0] = container->getColor(p).r;
                  result[offset + 1] = container->getColor(p).g;
                  result[offset + 2] = container->getColor(p).b;
              }
          }
      }
      stbi_write_bmp("result.bmp", container->width(), container->height(), 3, result);
  }

private:
  bool belongsTo(Pixel p, Pixel o, int w, int h) {
      return p.x >= o.x && p.x < o.x+w && p.y >= o.y && p.y < o.y+h;
  }

};

__global__ void getWindowDiff(uc *imgContainer, int containerWidth,
                              uc *imgObject, int objectWidth, int objectHeight,
                              uc *resultMatrix, unsigned int step)
{
    if(threadIdx.x != 0) return;

    int ox = blockIdx.x * (blockDim.x + step);// + threadIdx.x;
    int oy = blockIdx.y * (blockDim.y + step);// + threadIdx.y;

    int totalDiff = 0;
    for(int y = 0; y < objectHeight; ++y)
    {
        for(int x = 0; x < objectWidth; ++x)
        {
            totalDiff += abs(imgContainer[y * objectWidth + x] - imgObject[y * objectWidth + x]);
        }
    }

    resultMatrix[oy * containerWidth + ox] = totalDiff / (objectWidth * objectHeight);
}

void CheckCudaError(char sms[], int line);

int main(int argc, char** argv)
{
  if(argc < 4)
      cout << "usage: " << argv[0] << " <container file name> <object file name>" << endl;

  FaceDetection fc(argv[1], argv[2]);

  unsigned int nThreads = 1024;
  unsigned int nBlocks = 65535; //SUPONIENDO QUE LAS DOS IMAGENES SON CUADRADAS!
  unsigned int step = (fc.container->width() - fc.object->width()) / nBlocks;


  //Save two images in grayscale

  int numBytesContainer = fc.container->width() * fc.container->height() * sizeof(uc);
  int numBytesObject = fc.object->width() * fc.object->height() * sizeof(uc);
  uc *h_resultDiffMatrix = (uc*) malloc(numBytesContainer);
  uc *h_containerImageGS = (uc*) malloc(numBytesContainer);
  uc *h_objectImageGS = (uc*) malloc(numBytesObject);

  //Fill resultDiffMatrix with zeroes
  for(int y = 0; y < fc.container->height(); ++y) {
      for(int x = 0; x < fc.container->width(); ++x) {
          h_objectImageGS[y * fc.container->width() + x] = 0;
      }
  }

  //Fill container image with its grayscale
  for(int y = 0; y < fc.container->height(); ++y) {
      for(int x = 0; x < fc.container->width(); ++x) {
          h_containerImageGS[y * fc.container->width() + x] = fc.container->getGrayScale(Pixel(x,y));
      }
  }

  //Fill object image with its grayscale
  for(int y = 0; y < fc.object->height(); ++y) {
      for(int x = 0; x < fc.object->width(); ++x) {
          h_objectImageGS[y * fc.object->width() + x] = fc.object->getGrayScale(Pixel(x,y));
      }
  }

  // ////////////////////////////////////////


  //Obtiene Memoria [pinned] en el host
  //hipHostMalloc((float**)&h_x, numBytes);
  //hipHostMalloc((float**)&h_y, numBytes);
  //hipHostMalloc((float**)&H_y, numBytes);   // Solo se usa para comprobar el resultado


  // Obtener Memoria en el device

  //For every pixel(x,y), it contains the result of the avg diff of the window beginning in that pixel
  uc *d_resultDiffMatrix;
  uc *d_containerImageGS; //bigger image
  uc *d_objectImageGS;    //smaller image

  hipMalloc((uc**)&d_resultDiffMatrix, numBytesContainer); //result diff matrix
  hipMalloc((uc**)&d_containerImageGS, numBytesContainer);
  hipMalloc((uc**)&d_objectImageGS, numBytesObject);
  CheckCudaError((char *) "Obtener Memoria en el device", __LINE__);

  // Copiar datos desde el host en el device
  hipMemcpy(d_resultDiffMatrix, h_resultDiffMatrix,
             numBytesContainer, hipMemcpyHostToDevice);
  hipMemcpy(d_containerImageGS, h_containerImageGS,
             numBytesContainer, hipMemcpyHostToDevice);
  hipMemcpy(d_objectImageGS, h_objectImageGS,
             numBytesObject, hipMemcpyHostToDevice);
  CheckCudaError((char *) "Copiar Datos Host --> Device", __LINE__);

  // ////////////////////////////////////////

  // Ejecutar el kernel
  getWindowDiff<<<nBlocks, nThreads>>>(d_containerImageGS, fc.container->width(),
                                       d_objectImageGS, fc.object->width(), fc.object->height(),
                                       d_resultDiffMatrix, step);
  CheckCudaError((char *) "Invocar Kernel", __LINE__);


  // Obtener el resultado desde el host
  // Guardamos el resultado en H_y para poder comprobar el resultado
  hipMemcpy(h_resultDiffMatrix, d_resultDiffMatrix, numBytesContainer, hipMemcpyDeviceToHost);
  CheckCudaError((char *) "Copiar Datos Device --> Host", __LINE__);

  // Liberar Memoria del device
  hipFree(d_resultDiffMatrix);
  hipFree(d_containerImageGS);
  hipFree(d_objectImageGS);

  hipDeviceSynchronize();

  printf("nThreads: %d\n", nThreads);
  printf("nBlocks: %d\n", nBlocks);
}


void CheckCudaError(char sms[], int line) {
  hipError_t error;
  error = hipGetLastError();
  if (error) {
    printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
    exit(EXIT_FAILURE);
  }
}


